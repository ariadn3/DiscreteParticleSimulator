#include "hip/hip_runtime.h"
#include "init.h"

// Randomly generates an array of particles
__host__ void randomiseParticles(particle_t* particleArray, int slowFactor, int n,
        float L, float r) {
    float* posArray = generatePosition(n, L, r);
    float* veloArray = generateVelocity(slowFactor, n, L, r);
    
    for (int i = 0; i < n; i++) {
        particleArray[i].id = i;
        particleArray[i].x = posArray[2 * i];
        particleArray[i].y = posArray[2 * i + 1];
        particleArray[i].v_x = veloArray[2 * i];
        particleArray[i].v_y = veloArray[2 * i + 1];
        particleArray[i].w_collisions = 0;
        particleArray[i].p_collisions = 0;
    }

    free(posArray);
    free(veloArray);
}

// Generates an array of non-overlapping position values
__host__ float* generatePosition(int n, float L, float r) {
    static float* posArray;
    posArray = (float*) malloc(n * 2 * sizeof(float));
    srand(SEED);
    
    // Checks pre-conditions (read our report)
    if (L < 2 * r) {
        printf("Assumption violated: L < (2 * r)\n");
        exit(1);
    } else if (n * r * r > L * L) {
        printf("Assumption violated: (n * r * r) > (L * L)\n");
        exit(1);
    }
    
    // Bounds for generating positions
    float minPos = r, maxPos = L - r, posRange = maxPos - minPos;
    for (int i = 0; i < n; i++) {
        while (true) {
            posArray[2 * i] = minPos + posRange * (rand() / (float)RAND_MAX);
            posArray[2 * i +1] = minPos + posRange * (rand() / (float)RAND_MAX);
            bool flag = true;
            for (int j = 0; j < i; j++) {
                if (2 * r > sqrt(pow(posArray[2 * i] - posArray[2 * j], 2)
                            + pow(posArray[2 * i + 1] - posArray[2 * j + 1], 2))) {
                    flag = false;
                    break;
                }
            }
            if (flag) break;
        }
    }

    return posArray;
}

// Generates an array of velocity values
__host__ float* generateVelocity(int slowFactor, int n, float L, float r) {
    static float* veloArray;
    veloArray = (float*) malloc(n * 2 * sizeof(float));
    
    // Bounds for generating velocities
    float minVelocity = L / (8 * r * slowFactor);
    float maxVelocity = L / (4 * slowFactor);
    float velocityRange = maxVelocity - minVelocity;
    float minPolarAngle = 0, maxPolarAngle = 2 * M_PI;
    float angleRange = maxPolarAngle - minPolarAngle;
    float v, theta;

    for (int i = 0; i < n; i++) {
        v = minVelocity + velocityRange * (rand() / (float)RAND_MAX);
        theta = minPolarAngle + angleRange * (rand() / (float)RAND_MAX);
        veloArray[2 * i] = v * cos(theta);
        veloArray[2 * i + 1] = v * sin(theta);
    }
    return veloArray;
}

