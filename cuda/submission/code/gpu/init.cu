#include "hip/hip_runtime.h"
#include "init.h"

// Randomly generates an array of particles
__host__ void randomiseParticles(particle_t* particleArray, int slowFactor, int n,
        double L, double r) {
    double* posArray = generatePosition(n, L, r);
    double* veloArray = generateVelocity(slowFactor, n, L, r);
    
    for (int i = 0; i < n; i++) {
        particleArray[i].id = i;
        particleArray[i].x = posArray[2 * i];
        particleArray[i].y = posArray[2 * i + 1];
        particleArray[i].v_x = veloArray[2 * i];
        particleArray[i].v_y = veloArray[2 * i + 1];
        particleArray[i].w_collisions = 0;
        particleArray[i].p_collisions = 0;
    }

    free(posArray);
    free(veloArray);
}

// Generates an array of non-overlapping position values
__host__ double* generatePosition(int n, double L, double r) {
    static double* posArray;
    posArray = (double*) malloc(n * 2 * sizeof(double));
    srand(SEED);
    
    // Checks pre-conditions (read our report)
    if (L < 2 * r) {
        printf("Assumption violated: L < (2 * r)\n");
        exit(1);
    } else if (n * r * r > L * L) {
        printf("Assumption violated: (n * r * r) > (L * L)\n");
        exit(1);
    }
    
    // Bounds for generating positions
    double minPos = r, maxPos = L - r, posRange = maxPos - minPos;
    for (int i = 0; i < n; i++) {
        while (true) {
            posArray[2 * i] = minPos + posRange * (rand() / (double)RAND_MAX);
            posArray[2 * i +1] = minPos + posRange * (rand() / (double)RAND_MAX);
            bool flag = true;
            for (int j = 0; j < i; j++) {
                if (2 * r > sqrt(pow(posArray[2 * i] - posArray[2 * j], 2)
                            + pow(posArray[2 * i + 1] - posArray[2 * j + 1], 2))) {
                    flag = false;
                    break;
                }
            }
            if (flag) break;
        }
    }

    return posArray;
}

// Generates an array of velocity values
__host__ double* generateVelocity(int slowFactor, int n, double L, double r) {
    static double* veloArray;
    veloArray = (double*) malloc(n * 2 * sizeof(double));
    
    // Bounds for generating velocities
    double minVelocity = L / (8 * r * slowFactor);
    double maxVelocity = L / (4 * slowFactor);
    double velocityRange = maxVelocity - minVelocity;
    double minPolarAngle = 0, maxPolarAngle = 2 * M_PI;
    double angleRange = maxPolarAngle - minPolarAngle;
    double v, theta;

    for (int i = 0; i < n; i++) {
        v = minVelocity + velocityRange * (rand() / (double)RAND_MAX);
        theta = minPolarAngle + angleRange * (rand() / (double)RAND_MAX);
        veloArray[2 * i] = v * cos(theta);
        veloArray[2 * i + 1] = v * sin(theta);
    }
    return veloArray;
}

