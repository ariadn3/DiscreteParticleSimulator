#include <math.h>

#include "init.h"
#include "kernels.h"
#include "structs.h"

#define DEBUG_LEVEL 0
#define SLOW_FACTOR 1
#define NO_COLLISION 2

__host__ void simulate();
__host__ void printAll(bool, int, int, particle_t**);
__host__ void resolveValidCollisions(collision_t**, int*, double, double);
__host__ void filterCollisions(collision_t**, bool*, int*);
__host__ int cmpCollision(const void*, const void*);

hipError_t allocStatus;

// Shared simulation parameters
__constant__ int n, s;
__constant__ double l, r;

// Shared data
__managed__ int* numCollisions;
__managed__ particle_t** ps;
__managed__ bool* states;
__managed__ collision_t** cs;

__host__ void assertMallocSuccess(char* buff) {
    if (allocStatus != hipSuccess) {
        printf("Failed to dynamically allocate memory for %s\n", buff);
        printf("%s\n", hipGetErrorString(allocStatus));
        exit(1);
    }
}

__host__ int main(int argc, char** argv) {
    int hostN, hostL, hostR, hostS;
    bool willPrint;

    // Read in N, L, r, S and finally simulation mode
    scanf("%d\n%lf\n%lf\n%d\n", &hostN, &hostL, &hostR, &hostS);
    char* buffer = (char*) malloc(sizeof(char) * 140);
    scanf("%s\n", buffer);

    // Determine if this simulation will run in 'print' or 'perf' mode
    if(strcmp(buffer, "print") == 0) {
        willPrint = true;
    } else if (strcmp(buffer, "perf") == 0) {
        willPrint = false;
    } else {
        printf("Neither 'print' or 'perf' words are present. Exiting...\n");
        exit(1);
    }
    
    // Determine if there is a need to randomise particles
    int i;
    double x, y, v_x, v_y;
    bool isInitialised = false;
    allocStatus = hipMallocManaged((void**) &ps, hostN * sizeof(particle_t*));
    assertMallocSuccess("particle_t** ps");

    // If initial positions and velocities of particles are provided, read them
    while (fgets(buffer, 140, stdin) != EOF) {
        isInitialised = true;
        sscanf(buffer, "%d %lf %lf %lf %lf", &i, &x, &y, &v_x, &v_y);
        particles[i] = build_particle(i, x, y, v_x / slowFactor, v_y / slowFactor);
    }

    // Otherwise randomise the initial positions and velocities
    if (!isInitialised) randomiseParticles(particles, slowFactor, p->n, p->l, p->r);
    free(buffer);

    // Copy to GPU constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(n), &hostN, sizeof(n));
    hipMemcpyToSymbol(HIP_SYMBOL(l), &hostL, sizeof(l));
    hipMemcpyToSymbol(HIP_SYMBOL(r), &hostR, sizeof(r));
    hipMemcpyToSymbol(HIP_SYMBOL(s), &hostS, sizeof(s));

    // Initialise global collision counter
    allocStatus = hipMallocManaged((void**) &numCollisions, sizeof(int));
    assertMallocSuccess("int* numCollisions");

    // Initialise global particle collision state array
    allocStatus = hipMallocManaged((void**) &states, hostN * sizeof(bool));
    assertMallocSuccess("bool* states");

    for (int i = 0; i < hostN; i++) {
        states[i] = false;
    }
    
    // Initialise global collisions array - keep up to 8N collision candidates
    allocStatus = hipMallocManaged((void**) &cs, 8 * hostN * sizeof(collision_t*));
    assertMallocSuccess("collision_t** cs");

    simulate();
    
    return 0;
}

__host__ void simulate() {
    // Unconditionally print the starting state of the simulation
    printAll(false, hostN, 0, ps);
    
    for (int step = 1; step <= s; step++) {
        *numCollisions = 0;

        // ===== CHECKING AND ADDING COLLISION CANDIDATES =====
        for (int p = 0; p < n; p++) {
            double wallTime = checkWallCollision(r, l, ps[p]);
            if (wallTime != NO_COLLISION) {
                collision_t* candidate = build_collision(ps[p], NULL, wallTime);
                // #pragma CS
                cs[*numCollisions] = candidate;
                (*numCollisions)++;
                // #end CS
            }

            for (int q = p + 1; q < n; q++) {
                double time = checkCollision(r, ps[p], ps[q]);

                if (time != NO_COLLISION) {
                    collision_t* candidate = build_collision(ps[p], ps[q], time);
                    // #pragma CS
                    cs[*numCollisions] = candidate;
                    (*numCollisions)++;
                    // #end CS
                }
            }
        }

        // ===== FILTER COLLISION CANDIDATES TO VALID COLLISION =====
        filterCollisions(cs, states, numCollisions);
        
        // ===== RESOLVE VALID COLLISIONS =====
        resolveValidCollisions(cs, numCollisions, l, r);

        updateParticles(ps, n, states);

        // ===== PRINT SIMULATION DETAILS =====
        if (step == s) printAll(true, n, step, ps);
        else if (willPrint) printAll(false, n, step, ps);
    }
    
    return 0;
}

__host__ void printAll(bool includeCollisions, int n, int step,
        particle_t** particles) {
    // Parallelise this
    for (int i = 0; i < n; i++) {
        char* details;
        if (includeCollisions) {
            details = particle_string_full(particles[i]);
        } else {
            details = particle_string(particles[i]);
        }
        printf("%d %s", step, details);
        free(details);
    }
}

// Filters the collisions according to the time that it took place
__host__ void filterCollisions(collision_t** collisionArray, bool* hasCollided,
        int* numCollisions) {
    // Quicksort all collision candidates with the comparator function
    qsort(collisionArray, *numCollisions, sizeof(collision_t*), &cmpCollision);

    int saveIndex = 0;
    collision_t* curCollision;
    for (int curIndex = 0; curIndex < *numCollisions; curIndex++) {
        curCollision = collisionArray[curIndex];
        
        // printf("=== Particle %d and %d collided ===\n", curCollision->p->id,
        //         curCollision->q == NULL ? -1 : curCollision->q->id);
        if (hasCollided[curCollision->p->id]
                || (curCollision->q != NULL && hasCollided[curCollision->q->id])) {
            // Particle p has already collided OR particle q has already collided
            // -> discard this colision candidate
            free_collision(curCollision);
        } else {
            // Collision candidate is valid - marked p, q as collided
            hasCollided[curCollision->p->id] = true;

            if (curCollision-> q != NULL) hasCollided[curCollision->q->id] = true;
            // Re-use collision candidates array to store valid collisions
            collisionArray[saveIndex] = collisionArray[curIndex];
            saveIndex++;
        }
    }

    *numCollisions = saveIndex;
}

// Comparator for sorting collisions, earlier time then smaller particle 'p' id
__host__ int cmpCollision(const void* collisionA, const void* collisionB) {
    collision_t* firstCollision = *(collision_t**) collisionA;
    collision_t* secondCollision = *(collision_t**) collisionB;
    
    if (firstCollision->time == secondCollision->time) {
        // If both collisions involve the same first particle
        // Then prioritize wall collision, otherwise prioritize lower 2nd particle ID
        if (firstCollision->p->id == secondCollision->p->id) {
            if (firstCollision->q == NULL) return -1;
            else if (secondCollision->q == NULL) return 1;
            else return (firstCollision->q->id < secondCollision->q->id) ? -1 : 1;
        }
        // If two collisions occur at exactly the same time
        // Then prioritise the one which involves the particle P with lower ID
        return (firstCollision->p->id < secondCollision->p->id) ? -1 : 1;
    } else {
        // Otherwise prioritise the collision occurring at an earlier time
        return (firstCollision->time < secondCollision->time) ? -1 : 1;
    }
}

// Updates particles involved in all valid collisions in collision array
__host__ void resolveValidCollisions(collision_t** collisionArray, int* numCollisions,
        double L, double r) {
    collision_t* curCollision;
    for (int i = 0; i < *numCollisions; i++) {
        curCollision = collisionArray[i];
        settleCollision(curCollision, L, r);
        free_collision(curCollision);
    }
}

