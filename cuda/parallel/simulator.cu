#include "hip/hip_runtime.h"
#include <math.h>

#include "init.h"
#include "structs.h"

#define DEBUG_LEVEL 0
#define SLOW_FACTOR 1
#define NO_COLLISION 2
#define EDGE_TOLERANCE 1e-14

__host__ void simulate();
__host__ void printAll(bool, int, int, particle_t*);
__host__ void resolveValidCollisions(collision_t*, int*, double, double);
__host__ void filterCollisions(collision_t*, bool*, int*);
__host__ int cmpCollision(const void*, const void*);

__global__ void checkWallCollision(double, double, particle_t*);
__global__ void checkCollision(double, particle_t*, particle_t*);
__global__ void updateParticles(particle_t**, int, bool*);
__global__ void settleCollision(collision_t*, double, double);

int hostN, hostS;
double hostL, hostR;
bool willPrint;

hipError_t allocStatus;

// Shared simulation parameters
__constant__ int n, s;
__constant__ double l, r;

// Shared data
__managed__ int numCollisions;
__managed__ particle_t* ps;
__managed__ bool* states;
__managed__ collision_t* cs;

__host__ void assertMallocSuccess(char* buff) {
    if (allocStatus != hipSuccess) {
        printf("Failed to dynamically allocate memory for %s\n", buff);
        printf("%s\n", hipGetErrorString(allocStatus));
        exit(1);
    }
}

__host__ int main(int argc, char** argv) {
    int hostN, hostL, hostR, hostS;
    bool willPrint;

    // Read in N, L, r, S and finally simulation mode
    scanf("%d\n%lf\n%lf\n%d\n", &hostN, &hostL, &hostR, &hostS);
    char* buffer = (char*) malloc(sizeof(char) * 140);
    scanf("%s\n", buffer);

    // Determine if this simulation will run in 'print' or 'perf' mode
    if(strcmp(buffer, "print") == 0) {
        willPrint = true;
    } else if (strcmp(buffer, "perf") == 0) {
        willPrint = false;
    } else {
        printf("Neither 'print' or 'perf' words are present. Exiting...\n");
        exit(1);
    }
    
    // Determine if there is a need to randomise particles
    int i;
    double x, y, v_x, v_y;
    bool isInitialised = false;
    allocStatus = hipMallocManaged((void**) &ps, hostN * sizeof(particle_t));
    assertMallocSuccess("particle_t** ps");

    // If initial positions and velocities of particles are provided, read them
    while (fgets(buffer, 140, stdin) != EOF) {
        isInitialised = true;
        sscanf(buffer, "%d %lf %lf %lf %lf", &i, &x, &y, &v_x, &v_y);
        particles[i] = build_particle(i, x, y, v_x / slowFactor, v_y / slowFactor);
    }

    // Otherwise randomise the initial positions and velocities
    if (!isInitialised) randomiseParticles(particles, slowFactor, p.n, p.l, p.r);
    free(buffer);

    // Copy to GPU constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(n), &hostN, sizeof(n));
    hipMemcpyToSymbol(HIP_SYMBOL(l), &hostL, sizeof(l));
    hipMemcpyToSymbol(HIP_SYMBOL(r), &hostR, sizeof(r));
    hipMemcpyToSymbol(HIP_SYMBOL(s), &hostS, sizeof(s));

    // Initialise global collision counter
    allocStatus = hipMallocManaged((void**) &numCollisions, sizeof(int));
    assertMallocSuccess("int* numCollisions");

    // Initialise global particle collision state array
    allocStatus = hipMallocManaged((void**) &states, hostN * sizeof(bool));
    assertMallocSuccess("bool* states");

    for (int i = 0; i < hostN; i++) {
        states[i] = false;
    }
    
    // Initialise global collisions array - keep up to 8N collision candidates
    allocStatus = hipMallocManaged((void**) &cs, 8 * hostN * sizeof(collision_t));
    assertMallocSuccess("collision_t** cs");

    simulate();
    
    return 0;
}

__host__ void simulate() {
    // Unconditionally print the starting state of the simulation
    printAll(false, hostN, 0, ps);
    
    int pwChunkSize = 32;
    dim3 pwGrid((hostN+pwChunkSize-1)/pwChunkSize);
    dim3 pwThread(pwChunkSize);

    int ppChunkSize = 32
    dim3 ppGrid((hostN+1)/2, (hostN+ppChunkSize-1)/ppChunkSize);
    dim3 ppThread(pwChunkSize);

    int resolveChunkSize = 32;

    int updateChunkSize = 32;
    dim3 updateGrid((hostN+updateChunkSize-1)/updateChunkSize);
    dim3 updateThread(updateChunkSize);

    for (int step = 1; step <= s; step++) {
        numCollisions = 0;

        // ===== CHECKING AND ADDING COLLISION CANDIDATES =====
        checkWallCollision<<<pwGrid, pwThread>>();
        settleCollision<<<ppGrid, ppChunkSize>>>();
        hipDeviceSynchronize();

        // ===== FILTER COLLISION CANDIDATES TO VALID COLLISION =====
        filterCollisions(cs, states, numCollisions);
        
        // ===== RESOLVE VALID COLLISIONS =====
        dim3 resolveGrid((numCollisions+resolveChunkSize-1)/resolveChunkSize);
        dim3 resolveThread(resolveChunkSize);

        settleCollision<<<resolveGrid, resolveThread>>>();
        updateParticles<<<updateGrid, updateThread>>>();
        hipDeviceSynchronize();

        // ===== PRINT SIMULATION DETAILS =====
        if (step == s) printAll(true, n, step, ps);
        else if (willPrint) printAll(false, n, step, ps);
    }
    
    return 0;
}

__host__ void printAll(bool includeCollisions, int step, particle_t* particles)
{
    for (int i = 0; i < hostN; i++) {
        char* details;
        if (includeCollisions) {
            details = particle_string_full(&particles[i]);
        } else {
            details = particle_string(&particles[i]);
        }
        printf("%d %s", step, details);
        free(details);
    }
}

// Filters the collisions according to the time that it took place
__host__ void filterCollisions(collision_t* collisionArray, bool* hasCollided) {
    // Quicksort all collision candidates with the comparator function
    qsort(collisionArray, numCollisions, sizeof(collision_t), cmpCollision);

    int saveIndex = 0;
    collision_t curCollision;

    // Walk down collision array and retain valid collisions
    for (int curIndex = 0; curIndex < numCollisions; curIndex++) {
        curCollision = collisionArray[curIndex];
        
        if (hasCollided[curCollision.p.id]
                || (curCollision.q != NULL && hasCollided[curCollision.q.id])) {
            // Particle p has already collided OR particle q has already collided
            // -> discard this colision candidate
            // DO NOTHING (allow this struct to be overwritten later)
        } else {
            // Collision candidate is valid - marked p, q as collided
            hasCollided[curCollision.p.id] = true;

            if (curCollision. q != NULL) hasCollided[curCollision.q.id] = true;
            // Re-use collision candidates array to store valid collisions
            collisionArray[saveIndex] = collisionArray[curIndex];
            saveIndex++;
        }
    }

    numCollisions = saveIndex;
}

// Comparator for sorting collisions, earlier time then smaller particle 'p' id
__host__ int cmpCollision(const void* collisionA, const void* collisionB) {
    collision_t firstCollision = *(collision_t*) collisionA;
    collision_t secondCollision = *(collision_t*) collisionB;
    
    if (firstCollision.time == secondCollision.time) {
        // If both collisions involve the same first particle
        // Then prioritize wall collision, otherwise prioritize lower 2nd particle ID
        if (firstCollision.p.id == secondCollision.p.id) {
            if (firstCollision.q == NULL) return -1;
            else if (secondCollision.q == NULL) return 1;
            else return (firstCollision.q.id < secondCollision.q.id) ? -1 : 1;
        }
        // If two collisions occur at exactly the same time
        // Then prioritise the one which involves the particle P with lower ID
        return (firstCollision.p.id < secondCollision.p.id) ? -1 : 1;
    } else {
        // Otherwise prioritise the collision occurring at an earlier time
        return (firstCollision.time < secondCollision.time) ? -1 : 1;
    }
}


        // for (int p = 0; p < n; p++) {
        //     double wallTime = checkWallCollision(r, l, ps[p]);
        //     if (wallTime != NO_COLLISION) {
        //         collision_t* candidate = build_collision(ps[p], NULL, wallTime);
        //         // #pragma CS
        //         cs[*numCollisions] = candidate;
        //         (*numCollisions)++;
        //         // #end CS
        //     }

        //     for (int q = p + 1; q < n; q++) {
        //         double time = checkCollision(r, ps[p], ps[q]);

        //         if (time != NO_COLLISION) {
        //             collision_t* candidate = build_collision(ps[p], ps[q], time);
        //             // #pragma CS
        //             cs[*numCollisions] = candidate;
        //             (*numCollisions)++;
        //             // #end CS
        //         }
        //     }
        // }

__global__ void checkWallCollision() {
    int index = blockIdx.x*gridDim.x + threadIdx.x;
    if (index >= n)
        return;
    particle_t p = ps[index];

    // Collision times with vertical and horizontal walls
    double x_time = NO_COLLISION;
    double y_time = NO_COLLISION;

    double margin = r + EDGE_TOLERANCE;
    // Particle's position after 1 time step
    double x1 = p.x + p.v_x;
    double y1 = p.y + p.v_y;

    // Check if particle would intersect a vertical wall after 1 time step
    // If yes -> compute the time this would happen
    // Also check: if x-velocity is 0 but particle collides with wall
    // -> moving along horizontal wall -> don't try to divide by 0
    if (p.v_x != 0) {
        if (x1 < margin) {
            x_time = (p.x - r) / -(p.v_x); 
        } else if (x1 > l - margin) {
            x_time = (l - r - p.x) / (p.v_x);
        }
    }

    // Check if particle would intersect a horizontal wall after 1 time step
    // If yes -> compute the time this would happen
    // Also check: if y-velocity is 0 but particle collides with wall
    // -> moving along vertical wall -> don't try to divide by 0
    if (p.v_y != 0) {
        if (y1 < margin) {
            y_time = (p.y - r) / -(p.v_y);
        } else if (y1 > l - margin) {
            y_time = (l - r - p.y) / (p.v_y);
        }
    }

    // printf("%lf %lf %lf %lf\n", x_time, y_time, x1, y1);

    // Pick earlier of two times the particle would collide with a wall
    return x_time < y_time ? x_time : y_time;
}

__global__ void checkCollision() {
    int pIndex = blockIdx.x;
    int qIndex = blockDim.x*blockIdx.y + threadIdx.x;
    particle_t p, q;
    if (qIndex > pIndex) {
        p = ps[pIndex];
        q = ps[qIndex];
    }
    else if (gridDim.x%2 == 0 || pIndex != gridDim.x-1) {
        p = ps[n - 1 - qIndex];
        q = ps[n - 1 - pIndex];
    }
    else
        return;

    // Difference in X and Y positions and velocities of particles P, Q
    double dX = q.x - p.x;
    double dY = q.y - p.y;
    double dVx = q.v_x - p.v_x;
    double dVy = q.v_y - p.v_y;

    // 0 <= dT <= 1 is the fraction of a time step
    // A, B, C are the coefficients of the (dT)^2, dT and 0-th order terms in
    // the quadratic equation describing distance between particles P, Q at time dT
    double A = dVx * dVx + dVy * dVy;
    double B = 2 * (dX * dVx + dY * dVy);
    double C = dX * dX + dY * dY - 4 * r * r;

    double discriminant = B * B - 4 * A * C;

    if (discriminant <= 0) {
        return NO_COLLISION;
    }
    
    // Distance curve y = d(t) is concave up and intersects y = 2r at two points
    // First intersect (root) is at smaller dT and we only compute this

    // Possible that two particles are currently phasing through (i.e. d(0) < 2r)
    // since only 1 collision was computed per particle -> we ignore any first roots
    // that are dT < 0
    double dT = (-B - sqrt(discriminant)) / 2 / A;

    if (dT >= 0 && dT <= 1) {
        return dT;
    } else {
        return NO_COLLISION;
    }
}

// Moves particles involved in a collision to their rightful place after the timestep
__global__ void settleCollision() {
    int collIndex = blockIdx.x*blockDim.x + threadIdx.x;
    if (collIndex >= numCollisions)
        return;
    collision_t curCollision = cs[collIndex];

    // Particles A and B (null if wall collision) in this collision
    particle_t A = curCollision.p;
    particle_t B = curCollision.q;
    double time = curCollision.time;

    // Advance A by the fractional time step dT until collision occurs
    A.x += time * A.v_x;
    A.y += time * A.v_y;

    // If the collision is against the wall, toggle directions
    if (B == NULL) {
        // Add to wall collision counter of A
        A.w_collisions += 1;
        // printf("Step %.14lf: particle %d collided with wall\n", time, A.id);
        if (A.x <= r + EDGE_TOLERANCE || A.x >= L - r - EDGE_TOLERANCE)
            A.v_x *= -1;
        if (A.y <= r + EDGE_TOLERANCE || A.y >= L - r - EDGE_TOLERANCE)
            A.v_y *= -1;
    }
    // If collision is against another particle
    else {
        // Add to particle collision counters of both A and B
        A.p_collisions += 1;
        B.p_collisions += 1;
        // printf("Step %.14lf: particle %d collided with particle %d\n",
        //        time, A.id, B.id);
        // Advance B by dT until collision occurs
        B.x += time * B.v_x;
        B.y += time * B.v_y;

        // Compute distance between A, B
        double distance = sqrt(pow(B.x - A.x, 2) + pow(B.y - A.y, 2));

        // Compute normal and tangent unit vectors along x-, y-axes
        double n_x = (B.x - A.x) / distance;
        double n_y = (B.y - A.y) / distance;
        double t_x = -n_y;
        double t_y = n_x;

        // Compute new normal and tangent unit vectors for particles A, B
        double v_an = n_x * A.v_x + n_y * A.v_y;
        double v_at = t_x * A.v_x + t_y * A.v_y;
        double v_bn = n_x * B.v_x + n_y * B.v_y;
        double v_bt = t_x * B.v_x + t_y * B.v_y;

        // printf("n_x = %.14f, n_y = %.14f\n", n_x, n_y);
        // printf("t_x = %.14f, t_y = %.14f\n", t_x, t_y);
        // printf("v_an = %.14f, v_at = %.14f\n", v_an, v_at);
        // printf("v_bn = %.14f, v_bt = %.14f\n", v_bn, v_bt);

        // printf("Pre-collision velocities: %.14f, %.14f, %.14f, %.14f\n",
        //    A.v_x, A.v_y, B.v_x, B.v_y);

        // Update resultant velocities along x- and y-axes for particles A, B
        A.v_x = v_bn * n_x + v_at * t_x;
        A.v_y = v_bn * n_y + v_at * t_y;
        B.v_x = v_an * n_x + v_bt * t_x;
        B.v_y = v_an * n_y + v_bt * t_y;

        // printf("Post-collision velocities: %.14f, %.14f, %.14f, %.14f\n",
        //    A.v_x, A.v_y, B.v_x, B.v_y);

        // If particle B will collide against the wall, check when it will collide 
        // with the nearest wall and take that time
        double time_bx = 1 - time, time_by = 1 - time;
        if (B.v_x != 0) {
            if (B.x + time_bx * B.v_x < r) time_bx = -(B.x - r) / B.v_x;
            else if (B.x + time_bx * B.v_x > L - r)
                time_bx = (L - r - B.x) / B.v_x;
        }

        if (B.v_y != 0) {
            if (B.y + time_by * B.v_y < r) time_by = -(B.y - r) / B.v_y;
            else if (B.y + time_by * B.v_y > L - r)
                time_by = (L - r - B.y) / B.v_y;
        }

        // If B collides with two walls after colliding with A, take lesser of
        // two times
        double time_b = (time_bx < time_by) ? time_bx : time_by;

        B.x += time_b * B.v_x;
        B.y += time_b * B.v_y;
    }

    // If particle A will collide against the wall, check when it will collide
    // with the nearest wall and take that time
    double time_ax = 1 - time, time_ay = 1 - time;
    if (A.v_x != 0) {
        if (A.x + time_ax * A.v_x < r) time_ax = -(A.x - r) / A.v_x;
        else if (A.x + time_ax * A.v_x > L - r) time_ax = (L - r - A.x) / A.v_x;
    }

    if (A.v_y != 0) {
        if (A.y + time_ay * A.v_y < r) time_ay = -(A.y - r)/ A.v_y;
        else if (A.y + time_ay * A.v_y > L - r) time_ay = (L - r - A.y) / A.v_y;
    }

    // If A collides with another wall after colliding, take lesser of two times
    double time_a = (time_ax < time_ay) ? time_ax : time_ay;

    A.x += time_a * A.v_x;
    A.y += time_a * A.v_y;
}

// Updates particles not involved in any collision
__global__ void updateParticles() {
    int index = blockIdx.x*gridDim.x + threadIdx.x;
    if (index >= n)
        return;

    particle_t curParticle = ps[index];
    if (!states[i]) {
        // Advance particle by its velocity
        curParticle.x += curParticle.v_x;
        curParticle.y += curParticle.v_y;
    } else {
        // Particle had collided -> reset its collision status for next step
        states[i] = false;
    }
}

