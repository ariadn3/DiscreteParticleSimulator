#include "hip/hip_runtime.h"
#include <math.h>

#include "init.h"
#include "structs.h"

#define SLOW_FACTOR 1
#define NO_COLLISION 2
#define EDGE_TOLERANCE 1e-14

__host__ void simulate();
__host__ void printAll(bool, int);
__host__ void resolveValidCollisions(collision_t*, int*, double, double);
__host__ void filterCollisions();
__host__ int cmpCollision(const void*, const void*);

__global__ void checkWallCollision();
__global__ void checkCollision();
__global__ void updateParticles();
__global__ void settleCollision();

int hostN, hostS;
double hostL, hostR;
bool willPrint;

hipError_t allocStatus;

// Shared simulation parameters
__constant__ int n, s;
__constant__ double l, r;

// Shared data
__managed__ int numCollisions;
__managed__ particle_t* ps;
__managed__ bool* states;
__managed__ collision_t* cs;

__host__ void assertMallocSuccess(char* buff) {
    if (allocStatus != hipSuccess) {
        printf("Failed to dynamically allocate memory for %s\n", buff);
        printf("%s\n", hipGetErrorString(allocStatus));
        exit(1);
    }
}

__host__ int main(int argc, char** argv) {
    // Read in N, L, r, S and finally simulation mode
    scanf("%d\n%lf\n%lf\n%d\n", &hostN, &hostL, &hostR, &hostS);
    char* buffer = (char*) malloc(sizeof(char) * 140);
    scanf("%s\n", buffer);

    // Determine if this simulation will run in 'print' or 'perf' mode
    if(strcmp(buffer, "print") == 0) {
        willPrint = true;
    } else if (strcmp(buffer, "perf") == 0) {
        willPrint = false;
    } else {
        printf("Neither 'print' or 'perf' words are present. Exiting...\n");
        exit(1);
    }
    
    // Determine if there is a need to randomise particles
    int i;
    double x, y, v_x, v_y;
    bool isInitialised = false;
    allocStatus = hipMallocManaged((void**) &ps, hostN * sizeof(particle_t));
    sprintf(buffer, "particle_t* ps");
    assertMallocSuccess(buffer);

    // If initial positions and velocities of particles are provided, read them
    while (fgets(buffer, 140, stdin) != NULL) {
        isInitialised = true;
        sscanf(buffer, "%d %lf %lf %lf %lf", &i, &x, &y, &v_x, &v_y);
        ps[i].id = i;
        ps[i].x = x;
        ps[i].y = y;
        ps[i].v_x = v_x / SLOW_FACTOR;
        ps[i].v_y = v_y / SLOW_FACTOR;
        ps[i].w_collisions = 0;
        ps[i].p_collisions = 0;
    }

    // Otherwise randomise the initial positions and velocities
    if (!isInitialised) randomiseParticles(ps, SLOW_FACTOR, hostN, hostL, hostR);

    // Copy to GPU constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(n), &hostN, sizeof(n));
    hipMemcpyToSymbol(HIP_SYMBOL(l), &hostL, sizeof(l));
    hipMemcpyToSymbol(HIP_SYMBOL(r), &hostR, sizeof(r));
    hipMemcpyToSymbol(HIP_SYMBOL(s), &hostS, sizeof(s));

    // Initialise global collision counter
    allocStatus = hipMallocManaged((void**) &numCollisions, sizeof(int));
    sprintf(buffer, "int numCollisions");
    assertMallocSuccess(buffer);

    // Initialise global particle collision state array
    allocStatus = hipMallocManaged((void**) &states, hostN * sizeof(bool));
    sprintf(buffer, "bool* states");
    assertMallocSuccess(buffer);

    for (int i = 0; i < hostN; i++) {
        states[i] = false;
    }
    
    // Initialise global collisions array - keep up to 8N collision candidates
    allocStatus = hipMallocManaged((void**) &cs, 8 * hostN * sizeof(collision_t));
    sprintf(buffer, "collision_t* cs");
    assertMallocSuccess(buffer);

    simulate();

    free(buffer);
    hipFree(&numCollisions);
    hipFree(ps);
    hipFree(states);
    hipFree(cs);
    
    return 0;
}

__host__ void simulate() {
    // Unconditionally print the starting state of the simulation
    printAll(false, 0);
    
    int pwChunkSize = 32;
    dim3 pwGrid((hostN + pwChunkSize - 1) / pwChunkSize);
    dim3 pwBlock(pwChunkSize);

    int ppChunkSize = 32;
    dim3 ppGrid((hostN + 1) / 2, (hostN + ppChunkSize - 1) / ppChunkSize);
    dim3 ppBlock(pwChunkSize);

    int resolveChunkSize = 32;

    int updateChunkSize = 32;
    dim3 updateGrid((hostN + updateChunkSize - 1) / updateChunkSize);
    dim3 updateBlock(updateChunkSize);

    for (int step = 1; step <= hostS; step++) {
        numCollisions = 0;

        // ===== CHECKING AND ADDING COLLISION CANDIDATES =====
        checkWallCollision<<<pwGrid, pwBlock>>>();

        hipDeviceSynchronize();
        
        // You know, we accidentally launched the settleCollision kernel here instead
        // of checkCollision and wondered why particles were colliding 3000 times in
        // 1 step - we wasted 2 hours on this :')
        checkCollision<<<ppGrid, ppBlock>>>();

        hipDeviceSynchronize();

        // ===== FILTER COLLISION CANDIDATES TO VALID COLLISION =====
        filterCollisions();
        
        hipDeviceSynchronize();
        
        // ===== RESOLVE VALID COLLISIONS =====
        dim3 resolveGrid((numCollisions + resolveChunkSize - 1) / resolveChunkSize);
        dim3 resolveBlock(resolveChunkSize);

        settleCollision<<<resolveGrid, resolveBlock>>>();
        
        hipDeviceSynchronize();

        updateParticles<<<updateGrid, updateBlock>>>();
        
        hipDeviceSynchronize();
        
        // ===== PRINT SIMULATION DETAILS =====
        if (step == hostS) printAll(true, step);
        else if (willPrint) printAll(false, step);

        hipDeviceSynchronize();
    }
}

__host__ void printAll(bool includeCollisions, int step) {
    for (int i = 0; i < hostN; i++) {
        char* details;
        if (includeCollisions) {
            details = particle_string_full(&ps[i]);
        } else {
            details = particle_string(&ps[i]);
        }
        printf("%d %s", step, details);
        free(details);
    }
}

// Filters the collisions according to the time that it took place
__host__ void filterCollisions() {
    // Quicksort all collision candidates with the comparator function
    qsort(cs, numCollisions, sizeof(collision_t), cmpCollision);

    int saveIndex = 0;
    collision_t curCollision;

    // Walk down collision array and retain valid collisions
    for (int curIndex = 0; curIndex < numCollisions; curIndex++) {
        curCollision = cs[curIndex];
        // printf("%s\n", collision_string(&curCollision));
        
        if (states[curCollision.p->id]
                || (curCollision.q != NULL && states[curCollision.q->id])) {
            // Particle p has already collided OR particle q has already collided
            // -> discard this colision candidate
            // DO NOTHING (allow this struct to be overwritten later)
        } else {
            // Collision candidate is valid - marked p, q as collided
            states[curCollision.p->id] = true;

            if (curCollision.q != NULL) states[curCollision.q->id] = true;
            // Re-use collision candidates array to store valid collisions
            cs[saveIndex] = cs[curIndex];
            saveIndex++;
        }
    }

    numCollisions = saveIndex;
}

// Comparator for sorting collisions, earlier time then smaller particle 'p' id
__host__ int cmpCollision(const void* collisionA, const void* collisionB) {
    collision_t firstCollision = *(collision_t*) collisionA;
    collision_t secondCollision = *(collision_t*) collisionB;
   
    if (firstCollision.time == secondCollision.time) {
        // If both collisions involve the same first particle
        // Then prioritize wall collision, otherwise prioritize lower 2nd particle ID
        if (firstCollision.p->id == secondCollision.p->id) {
            if (firstCollision.q == NULL) return -1;
            else if (secondCollision.q == NULL) return 1;
            else return (firstCollision.q->id < secondCollision.q->id) ? -1 : 1;
        }
        // If two collisions occur at exactly the same time
        // Then prioritise the one which involves the particle P with lower ID
        return (firstCollision.p->id < secondCollision.p->id) ? -1 : 1;
    } else {
        // Otherwise prioritise the collision occurring at an earlier time
        return (firstCollision.time < secondCollision.time) ? -1 : 1;
    }
}

__global__ void checkWallCollision() {
    int index = blockIdx.x * gridDim.x + threadIdx.x;
    
    if (index >= n)
        return;

    particle_t p = ps[index];

    // Collision times with vertical and horizontal walls
    double x_time = NO_COLLISION;
    double y_time = NO_COLLISION;

    double margin = r + EDGE_TOLERANCE;
    // Particle's position after 1 time step
    double x1 = p.x + p.v_x;
    double y1 = p.y + p.v_y;

    // Check if particle would intersect a vertical wall after 1 time step
    // If yes -> compute the time this would happen
    // Also check: if x-velocity is 0 but particle collides with wall
    // -> moving along horizontal wall -> don't try to divide by 0
    if (p.v_x != 0) {
        if (x1 < margin) {
            x_time = (p.x - r) / -(p.v_x); 
        } else if (x1 > l - margin) {
            x_time = (l - r - p.x) / (p.v_x);
        }
    }

    // Check if particle would intersect a horizontal wall after 1 time step
    // If yes -> compute the time this would happen
    // Also check: if y-velocity is 0 but particle collides with wall
    // -> moving along vertical wall -> don't try to divide by 0
    if (p.v_y != 0) {
        if (y1 < margin) {
            y_time = (p.y - r) / -(p.v_y);
        } else if (y1 > l - margin) {
            y_time = (l - r - p.y) / (p.v_y);
        }
    }

    // printf("%lf %lf %lf %lf\n", x_time, y_time, x1, y1);

    // Pick earlier of two times the particle would collide with a wall
    double wall_time = x_time < y_time ? x_time : y_time;
    
    if (wall_time != NO_COLLISION) {
        // atomicAdd returns the previous value of that address - we use this as a
        // ticket for this thread to write a collision to that specific index
        // Implicitly serves as a critical section
        int i = atomicAdd(&numCollisions, 1);
        // printf("CS%d: added by thread %d\n", i, index); 

        cs[i].p = &ps[p.id];
        cs[i].q = NULL;
        cs[i].time = wall_time;
    }
}

__global__ void checkCollision() {
    int pIndex = blockIdx.x;
    int qIndex = blockDim.x * blockIdx.y + threadIdx.x;

    // printf("Checking array computation (%d, %d)\n", pIndex, qIndex);

    particle_t p, q;
    
    // Ignore excess threads beyond the row of computation
    if (qIndex >= n) return;

    // Compute upper half of triangle
    if (qIndex > pIndex) {
        p = ps[pIndex];
        q = ps[qIndex];
    } else if (gridDim.x % 2 == 0 || pIndex != gridDim.x - 1) {
        // Compute reflected lower half of triangle folded to form a row
        p = ps[n - 1 - qIndex];
        q = ps[n - 1 - pIndex];
    } else {
        // Catch case where n is even -> odd number of rows of computation when folded
        // Ignore excess threads beyond the middle column
        return;
    }

    // Difference in X and Y positions and velocities of particles P, Q
    double dX = q.x - p.x;
    double dY = q.y - p.y;
    double dVx = q.v_x - p.v_x;
    double dVy = q.v_y - p.v_y;

    // 0 <= dT <= 1 is the fraction of a time step
    // A, B, C are the coefficients of the (dT)^2, dT and 0-th order terms in
    // the quadratic equation describing distance between particles P, Q at time dT
    double A = dVx * dVx + dVy * dVy;
    double B = 2 * (dX * dVx + dY * dVy);
    double C = dX * dX + dY * dY - 4 * r * r;

    double discriminant = B * B - 4 * A * C;

    if (discriminant <= 0) {
        return;
    }
    
    // Distance curve y = d(t) is concave up and intersects y = 2r at two points
    // First intersect (root) is at smaller dT and we only compute this

    // Possible that two particles are currently phasing through (i.e. d(0) < 2r)
    // since only 1 collision was computed per particle -> we ignore any first roots
    // that are dT < 0
    double dT = (-B - sqrt(discriminant)) / 2 / A;

    // Add a collision candidate if P, Q would collide during this time step
    if (dT >= 0 && dT <= 1) {
        // atomicAdd returns the previous value of that address - we use this as a
        // ticket for this thread to write a collision to that specific index
        // Implicitly serves as a critical section
        int i = atomicAdd(&numCollisions, 1);
        // printf("CS%d: p-p added by block %d thread %d\n", i, pIndex, qIndex); 
        
        cs[i].p = &ps[p.id];
        cs[i].q = &ps[q.id];
        cs[i].time = dT;
    }
}

// Moves particles involved in a collision to their rightful place after the timestep
__global__ void settleCollision() {
    int collIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (collIndex >= numCollisions)
        return;
    
    collision_t curCollision = cs[collIndex];

    // Particles A and B (null if wall collision) in this collision
    particle_t* __restrict__ A = curCollision.p;
    particle_t* __restrict__ B = curCollision.q;
    double time = curCollision.time;

    // Advance A by the fractional time step dT until collision occurs
    A->x += time * A->v_x;
    A->y += time * A->v_y;

    // If the collision is against the wall, toggle directions
    if (B == NULL) {
        // Add to wall collision counter of A
        A->w_collisions += 1;
        // printf("Step %.14lf: particle %d collided with wall\n", time, A->id);
        if (A->x <= r + EDGE_TOLERANCE || A->x >= l - r - EDGE_TOLERANCE)
            A->v_x *= -1;
        if (A->y <= r + EDGE_TOLERANCE || A->y >= l - r - EDGE_TOLERANCE)
            A->v_y *= -1;
    }
    // If collision is against another particle
    else {
        // Add to particle collision counters of both A and B
        A->p_collisions += 1;
        B->p_collisions += 1;
        // printf("Step %.14lf: particle %d collided with particle %d\n",
        //        time, A->id, B->id);
        // Advance B by dT until collision occurs
        B->x += time * B->v_x;
        B->y += time * B->v_y;

        // Compute distance between A, B
        double distance = sqrt(pow(B->x - A->x, 2) + pow(B->y - A->y, 2));

        // Compute normal and tangent unit vectors along x-, y-axes
        double n_x = (B->x - A->x) / distance;
        double n_y = (B->y - A->y) / distance;
        double t_x = -n_y;
        double t_y = n_x;

        // Compute new normal and tangent unit vectors for particles A, B
        double v_an = n_x * A->v_x + n_y * A->v_y;
        double v_at = t_x * A->v_x + t_y * A->v_y;
        double v_bn = n_x * B->v_x + n_y * B->v_y;
        double v_bt = t_x * B->v_x + t_y * B->v_y;

        // printf("n_x = %.14f, n_y = %.14f\n", n_x, n_y);
        // printf("t_x = %.14f, t_y = %.14f\n", t_x, t_y);
        // printf("v_an = %.14f, v_at = %.14f\n", v_an, v_at);
        // printf("v_bn = %.14f, v_bt = %.14f\n", v_bn, v_bt);

        // printf("Pre-collision velocities: %.14f, %.14f, %.14f, %.14f\n",
        //    A->v_x, A->v_y, B->v_x, B->v_y);

        // Update resultant velocities along x- and y-axes for particles A, B
        A->v_x = v_bn * n_x + v_at * t_x;
        A->v_y = v_bn * n_y + v_at * t_y;
        B->v_x = v_an * n_x + v_bt * t_x;
        B->v_y = v_an * n_y + v_bt * t_y;

        // printf("Post-collision velocities: %.14f, %.14f, %.14f, %.14f\n",
        //    A->v_x, A->v_y, B->v_x, B->v_y);

        // If particle B will collide against the wall, check when it will collide 
        // with the nearest wall and take that time
        double time_bx = 1 - time, time_by = 1 - time;
        if (B->v_x != 0) {
            if (B->x + time_bx * B->v_x < r) time_bx = -(B->x - r) / B->v_x;
            else if (B->x + time_bx * B->v_x > l - r)
                time_bx = (l - r - B->x) / B->v_x;
        }

        if (B->v_y != 0) {
            if (B->y + time_by * B->v_y < r) time_by = -(B->y - r) / B->v_y;
            else if (B->y + time_by * B->v_y > l - r)
                time_by = (l - r - B->y) / B->v_y;
        }

        // If B collides with two walls after colliding with A, take lesser of
        // two times
        double time_b = (time_bx < time_by) ? time_bx : time_by;

        B->x += time_b * B->v_x;
        B->y += time_b * B->v_y;
    }

    // If particle A will collide against the wall, check when it will collide
    // with the nearest wall and take that time
    double time_ax = 1 - time, time_ay = 1 - time;
    if (A->v_x != 0) {
        if (A->x + time_ax * A->v_x < r) time_ax = -(A->x - r) / A->v_x;
        else if (A->x + time_ax * A->v_x > l - r) time_ax = (l - r - A->x) / A->v_x;
    }

    if (A->v_y != 0) {
        if (A->y + time_ay * A->v_y < r) time_ay = -(A->y - r)/ A->v_y;
        else if (A->y + time_ay * A->v_y > l - r) time_ay = (l - r - A->y) / A->v_y;
    }

    // If A collides with another wall after colliding, take lesser of two times
    double time_a = (time_ax < time_ay) ? time_ax : time_ay;

    A->x += time_a * A->v_x;
    A->y += time_a * A->v_y;
}

// Updates particles not involved in any collision
__global__ void updateParticles() {
    int index = blockIdx.x * gridDim.x + threadIdx.x;
    if (index >= n)
        return;

    particle_t* curParticle = &ps[index];
    
    if (!states[index]) {
        // Advance particle by its velocity
        curParticle->x += curParticle->v_x;
        curParticle->y += curParticle->v_y;
    } else {
        // Particle had collided -> reset its collision status for next step
        states[index] = false;
    }
}

