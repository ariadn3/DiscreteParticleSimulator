#include "hip/hip_runtime.h"
#include "random.h"

// Randomly generates an array of particles
__host__ void randomiseParticles(particle_t** particleArray, int slowFactor, int n,
        double L, double r) {
    double* posArray = generatePosition(n, L, r);
    double* veloArray = generateVelocity(slowFactor, n, L, r);
    for (int i = 0; i < n; i++) {
        particleArray[i] = build_particle(i, posArray[2 * i], posArray[2 * i + 1],
                veloArray[2 * i], veloArray[2 * i + 1]);
    }
    free(posArray);
    free(veloArray);
}

// Generates an array of position values
__host__ double* generatePosition(int n, double L, double r) {
    static double* posArray;
    posArray = (double*) malloc(n * 2 * sizeof(double));
    srand(SEED);
    
    // Checks pre-conditions (read our report)
    if (L < 2 * r) {
        printf("Assumption violated: L < (2 * r)\n");
        exit(1);
    } else if (n * r * r > L * L) {
        printf("Assumption violated: (n * r * r) > (L * L)\n");
        exit(1);
    }
    
    // Bounds for generating positions
    double leftLimit = r, rightLimit = L - r, lenDiff = rightLimit - leftLimit;
    for (int i = 0; i < n; i++) {
        while (true) {
            posArray[2 * i] = leftLimit + lenDiff * (rand() / (double)RAND_MAX);
            posArray[2 * i +1] = leftLimit + lenDiff * (rand() / (double)RAND_MAX);
            bool flag = true;
            for (int j = 0; j < i; j++) {
                if (2 * r > sqrt(pow(posArray[2 * i] - posArray[2 * j], 2)
                            + pow(posArray[2 * i + 1] - posArray[2 * j + 1], 2))) {
                    flag = false;
                    break;
                }
            }
            if (flag) break;
        }
    }

    return posArray;
}

// Generates an array of velocity values
__host__ double* generateVelocity(int slowFactor, int n, double L, double r) {
    static double* veloArray;
    veloArray = (double*) malloc(n * 2 * sizeof(double));
    
    // Bounds for generating velocities
    double veloLeftLimit = L / (8 * r * slowFactor);
    double veloRightLimit = L / (4 * slowFactor);
    double veloDiff = veloRightLimit - veloLeftLimit;
    double angleLeftLimit = 0, angleRightLimit = 2 * M_PI;
    double angleDiff = angleRightLimit - angleLeftLimit;
    double velo, angle;

    for (int i = 0; i < n; i++) {
        velo = veloLeftLimit + veloDiff * (rand() / (double)RAND_MAX);
        angle = angleLeftLimit + angleDiff * (rand() / (double)RAND_MAX);
        veloArray[2 * i] = velo * cos(angle);
        veloArray[2 * i + 1] = velo * sin(angle);
    }
    return veloArray;
}

