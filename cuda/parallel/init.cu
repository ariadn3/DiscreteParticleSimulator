#include "hip/hip_runtime.h"
#include "init.h"

// Read in inputs from file and return initial simulation parameters as params_t
__host__ params_t* read_file(int slowFactor) {
    params_t* p = params();

    // Read in N, L, r, S and finally simulation mode
    scanf("%d\n%lf\n%lf\n%d\n", &(p->n), &(p->l), &(p->r), &(p->s));
    char* buffer = (char*) malloc(sizeof(char) * 140);
    scanf("%s\n", buffer);

    // Determine if this simulation will run in 'print' or 'perf' mode
    if(strcmp(buffer, "print") == 0) {
        p->willPrint = true;
    } else if (strcmp(buffer, "perf") == 0) {
        p->willPrint = false;
    } else {
        printf("Neither 'print' or 'perf' words are present. Exiting...\n");
        exit(1);
    }

    int i;
    double x, y, v_x, v_y;
    bool isInitialised = false;
    particle_t** particles = (particle_t**) malloc(p->n * sizeof(particle_t));

    // If initial positions and velocities of particles are provided, read them
    while (fgets(buffer, 140, stdin) != NULL) {
        isInitialised = true;
        sscanf(buffer, "%d %lf %lf %lf %lf", &i, &x, &y, &v_x, &v_y);
        particles[i] = build_particle(i, x, y, v_x / slowFactor, v_y / slowFactor);
    }

    // Otherwise randomise the initial positions and velocities
    if (!isInitialised) {
        randomiseParticles(particles, slowFactor, p->n, p->l, p->r);
    }

    p->particles = particles;
    return p;
}

__host__ void printAll(bool includeCollisions, int n, int step,
        particle_t** particles) {
    // Parallelise this
    for (int i = 0; i < n; i++) {
        char* details;
        if (includeCollisions) {
            details = particle_string_full(particles[i]);
        } else {
            details = particle_string(particles[i]);
        }
        printf("%d %s", step, details);
        free(details);
    }
}

// Randomly generates an array of particles
__host__ void randomiseParticles(particle_t** particleArray, int slowFactor, int n,
        double L, double r) {
    double* posArray = generatePosition(n, L, r);
    double* veloArray = generateVelocity(slowFactor, n, L, r);
    for (int i = 0; i < n; i++) {
        particleArray[i] = build_particle(i, posArray[2 * i], posArray[2 * i + 1],
                veloArray[2 * i], veloArray[2 * i + 1]);
    }
    free(posArray);
    free(veloArray);
}

// Generates an array of position values
__host__ double* generatePosition(int n, double L, double r) {
    static double* posArray;
    posArray = (double*) malloc(n * 2 * sizeof(double));
    srand(SEED);
    
    // Checks pre-conditions (read our report)
    if (L < 2 * r) {
        printf("Assumption violated: L < (2 * r)\n");
        exit(1);
    } else if (n * r * r > L * L) {
        printf("Assumption violated: (n * r * r) > (L * L)\n");
        exit(1);
    }
    
    // Bounds for generating positions
    double leftLimit = r, rightLimit = L - r, lenDiff = rightLimit - leftLimit;
    for (int i = 0; i < n; i++) {
        while (true) {
            posArray[2 * i] = leftLimit + lenDiff * (rand() / (double)RAND_MAX);
            posArray[2 * i +1] = leftLimit + lenDiff * (rand() / (double)RAND_MAX);
            bool flag = true;
            for (int j = 0; j < i; j++) {
                if (2 * r > sqrt(pow(posArray[2 * i] - posArray[2 * j], 2)
                            + pow(posArray[2 * i + 1] - posArray[2 * j + 1], 2))) {
                    flag = false;
                    break;
                }
            }
            if (flag) break;
        }
    }

    return posArray;
}

// Generates an array of velocity values
__host__ double* generateVelocity(int slowFactor, int n, double L, double r) {
    static double* veloArray;
    veloArray = (double*) malloc(n * 2 * sizeof(double));
    
    // Bounds for generating velocities
    double veloLeftLimit = L / (8 * r * slowFactor);
    double veloRightLimit = L / (4 * slowFactor);
    double veloDiff = veloRightLimit - veloLeftLimit;
    double angleLeftLimit = 0, angleRightLimit = 2 * M_PI;
    double angleDiff = angleRightLimit - angleLeftLimit;
    double velo, angle;

    for (int i = 0; i < n; i++) {
        velo = veloLeftLimit + veloDiff * (rand() / (double)RAND_MAX);
        angle = angleLeftLimit + angleDiff * (rand() / (double)RAND_MAX);
        veloArray[2 * i] = velo * cos(angle);
        veloArray[2 * i + 1] = velo * sin(angle);
    }
    return veloArray;
}

